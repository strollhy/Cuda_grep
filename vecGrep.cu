#include "hip/hip_runtime.h"
// Kernel definition
__global__ void vecGrep(char* buffer, 
			int* word_len, char* word, 
		        int* line_pos, int* line_len, 
		        int* find)
{
        // threadIdx.x is a built-in variable provided by CUDA at runtime
        int indx = threadIdx.x;
	int pos = line_pos[indx];
	int len = line_len[indx];
	
	for(int i=0; i<len; i++)
	{
		find[indx] = 1;

		for(int j=0; j<*word_len; j++) 
		{
        		if (buffer[pos+i+j] != word[j]) 
			{
        	        	find[indx] = 0;
				break;
			}
        	
		}
		
		if(find[indx] == 1)
			break;
	}
}


#include"grep.h"
#include<stdio.h>
#include<string.h>

void grep (char* buffer, char* word,
           int* line_pos, int* line_len)
{
        char *devPtrLine;
        char *devPtrWord;
	int *devPtrPos;
	int *devPtrLen;
	int *devPtrWlen;

        int *devPtrFind;
	int result[NUM];
	
	int buffer_size = buff_len * sizeof(char);
	int word_size = word_len * sizeof(char);
	int list_size = NUM * sizeof(int);
	
        hipMalloc((void**)&devPtrLine, buffer_size);
	hipMalloc((void**)&devPtrWlen, sizeof(int));
        hipMalloc((void**)&devPtrWord, word_size);
        hipMalloc((void**)&devPtrFind, list_size);
	hipMalloc((void**)&devPtrPos, list_size);
	hipMalloc((void**)&devPtrLen, list_size);

        hipMemcpy(devPtrLine, buffer, buffer_size, hipMemcpyHostToDevice);
	hipMemcpy(devPtrWlen, &word_len, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(devPtrWord, word, word_size, hipMemcpyHostToDevice);
	hipMemcpy(devPtrPos, line_pos, list_size, hipMemcpyHostToDevice);
	hipMemcpy(devPtrLen, line_len, list_size, hipMemcpyHostToDevice);

        // __global__ functions are called: Fun<<< Dg, Db, Ns >>>(parameter);
        vecGrep<<<1, NUM>>> (devPtrLine, 
				devPtrWlen, devPtrWord, 
				devPtrPos, devPtrLen, devPtrFind);
        hipMemcpy(result, devPtrFind, list_size, hipMemcpyDeviceToHost);

	// For display
	//for(int i=0; i<NUM; i++) printf("%d,", result[i]);
	//printf("\b \b\n");
      	
	/*	
	for(int i=0; i<NUM; i++)
		if(result[i])
		{ 
			int p = line_pos[i];
			int l = line_len[i];
			for(int j=0; j<l; j++)
				printf("%c", buffer[p+j]);
			printf("\n");
		}
	*/
	//printf(" %d ###################\n", segm_no);
	
        hipFree(devPtrLine);
	hipFree(devPtrWlen);
        hipFree(devPtrWord);
        hipFree(devPtrFind);
	hipFree(devPtrPos);
	hipFree(devPtrLen);
}

